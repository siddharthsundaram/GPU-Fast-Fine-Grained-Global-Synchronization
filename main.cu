#include "hip/hip_runtime.h"
#include "main.h"
#include "arg_parser.h"
#include <chrono>

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess)                                              \
        {                                                                      \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                           \
        }                                                                      \
    }                                                                          \

// __device__ int *d_locks;
// __device__ int *d_shared_data;
// __device__ int d_size;

int main(int argc, char **argv) {
    auto start = std::chrono::high_resolution_clock::now();
    char *input_file = argv[1]; 
    parse_args(argc, argv);
    
    FILE *file = fopen(input_file, "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening file %s\n", input_file);
        return 1;
    }
    
    int size = 0;
    fscanf(file, "%d", &size);
    
    int num_clients = 0;
    fscanf(file, "%d", &num_clients);
    
    int num_servers = 0;
    fscanf(file, "%d", &num_servers);
    
    fclose(file);

    // Sequential implementation
    if (seq) {
        sequential(size, num_clients);

    // Basic GPU implementation
    } else if (basic) {
        int *d_locks, *d_shared_data;
        CHECK(hipMalloc(&d_locks, size * sizeof(int)));
        CHECK(hipMalloc(&d_shared_data, size * sizeof(int)));
        CHECK(hipMemset(d_locks, 0, size * sizeof(int)));
        CHECK(hipMemset(d_shared_data, 0, size * sizeof(int)));
        dim3 block_size (std::min(num_clients, 256));
        dim3 grid_size ((num_clients + block_size.x - 1) / block_size.x + num_servers);
        basic_gpu<<<grid_size, block_size>>>(d_locks, d_shared_data, size, num_clients);
        hipDeviceSynchronize();

        int *h_shared_data = new int[size]();
        CHECK(hipMemcpy(h_shared_data, d_shared_data, size * sizeof(int), hipMemcpyDeviceToHost));

        // int total = 0;
        // for (int i = 0; i < size; i++) {
        //     printf("shared_data[%d] = %d\n", i, h_shared_data[i]); 
        //     total += h_shared_data[i];
        // }

        delete[] h_shared_data;
        CHECK(hipFree(d_shared_data));
        CHECK(hipFree(d_locks));

        // printf("TOTAL: %d\n", total);

    // Fine grain synch GPU implementation
    } else if (fg) {
        gpu_buffer(size, num_servers, num_clients);
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    printf("TIME: %lf\n", elapsed.count());

    return 0;
}

void gpu_buffer(int size, int num_servers, int num_clients) {
    int *h_locks = new int[size]();
    int *h_shared_data = new int[size](); 
    int *d_shared_data, *d_done;
    Buffer *d_bufs;
    CHECK(hipMalloc(&d_done, sizeof(int)));
    CHECK(hipMemset(d_done, 0, sizeof(int)));
    CHECK(hipMalloc(&d_bufs, num_servers * sizeof(Buffer)));
    CHECK(hipMemset(d_bufs, 0, num_servers * sizeof(Buffer)));
    CHECK(hipMalloc(&d_shared_data, size * sizeof(int)));
    CHECK(hipMemcpy(d_shared_data, h_shared_data, size * sizeof(int), hipMemcpyHostToDevice));

    dim3 block_size (std::min(num_clients, 256));
    dim3 grid_size ((num_clients + block_size.x - 1) / block_size.x + num_servers);
    // printf("NUM CLIENTS: %i\n", num_clients);
    // printf("BLOCK SIZE: %i\n", block_size);
    // printf("GRID SIZE: %i\n", grid_size); 
    int shared_mem_size = num_servers * size * sizeof(int); 
    counters_client_and_server_entry<<<4, 1024, shared_mem_size>>>(d_shared_data, size, num_servers, d_bufs, d_done, num_clients);

    CHECK(hipDeviceSynchronize());
    
    CHECK(hipMemcpy(h_shared_data, d_shared_data, size * sizeof(int), hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());
    
    // int total = 0;
    // for (int i = 0; i < size; i++) {
    //     printf("shared_data[%d] = %d\n", i, h_shared_data[i]); 
    //     total += h_shared_data[i];
    // }

    delete[] h_locks;
    delete[] h_shared_data;
    CHECK(hipFree(d_shared_data));
    CHECK(hipFree(d_done));
    CHECK(hipFree(d_bufs));

    // printf("TOTAL: %d\n", total);
}

__device__ bool try_lock(int data_id, int *locks) {
    return atomicCAS(&locks[data_id], 0, 1) == 0;
}

__device__ void unlock(int data_id, int *locks) {
    atomicExch(&locks[data_id], 0);
}

__device__ void critical_sec(int data_id, int *shared_data) {
    shared_data[data_id] += 1;
}

__global__ void basic_gpu(int *locks, int *shared_data, int size, int num_clients) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < num_clients) {
        int data_id = tid % size;

        bool success = false;
        do {
            if (try_lock(data_id, locks)) {
                critical_sec(data_id, shared_data);
                __threadfence();
                unlock(data_id, locks);
                success = true;
            }
        } while (!success);
    }
}

void sequential(int size, int num_increments) {
    int *res = new int[size]();
    for (int i = 0; i < num_increments; i++) {
        int j = i % size;
        res[j]++;
    }

    // int total = 0;
    // for (int i = 0; i < size; i++) {
    //     printf("sequential_data[%d] = %d\n", i, res[i]);
    //     total += res[i];
    // }

    // printf("TOTAL: %d\n", total);

    delete[] res;
}