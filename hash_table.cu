#include "hip/hip_runtime.h"
#include "hash_table.cuh"

namespace bpo = boost::program_options;

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess)                                              \
        {                                                                      \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                           \
        }                                                                      \
    }

// Default hash table parameters
#define HT_SIZE 1024                
#define MAX_LIST_NODES 10000000     
#define BLOCK_SIZE 256              

// Different collision factors for testing
#define CF_256 256
#define CF_1K 1024
#define CF_32K 32768
#define CF_128K 131072

double get_time() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration<double>(duration).count();
}

// Structure for hash table nodes (linked list)
struct Node {
    int key;
    int value;
    Node* next;
};

struct HashTable {
    Node** buckets;
    int size;
};

HashTable* create_hash_table(int size) {
    HashTable* table = (HashTable*)malloc(sizeof(HashTable));
    table->size = size;
    table->buckets = (Node**)malloc(sizeof(Node*) * size);
    
    for (int i = 0; i < size; i++) {
        table->buckets[i] = NULL;
    }
    
    return table;
}

void insert(HashTable* table, int key, int value) {
    int bucket = key % table->size;
    
    Node* new_node = (Node*)malloc(sizeof(Node));
    new_node->key = key;
    new_node->value = value;
    
    new_node->next = table->buckets[bucket];
    table->buckets[bucket] = new_node;
}

void free_hash_table(HashTable* table) {
    for (int i = 0; i < table->size; i++) {
        Node* current = table->buckets[i];
        while (current != NULL) {
            Node* temp = current;
            current = current->next;
            free(temp);
        }
    }
    
    free(table->buckets);
    free(table);
}

double run_sequential_benchmark(int pool_size) {
    printf("Running sequential benchmark with pool size %d\n", pool_size);
    
    HashTable* table = create_hash_table(HT_SIZE);
    
    // Create pool of elements for threads to randomly select from
    int* element_pool = (int*)malloc(sizeof(int) * pool_size);
    for (int i = 0; i < pool_size; i++) {
        element_pool[i] = rand();
    }
    
    double start_time = get_time();
    
    // Perform insertions - use pool_size as the number of operations
    for (int i = 0; i < pool_size; i++) {
        int element = element_pool[rand() % pool_size];
        insert(table, element, i);
    }
    
    double end_time = get_time();
    double elapsed_time = end_time - start_time;
    
    int total_nodes = 0;
    for (int i = 0; i < table->size; i++) {
        Node* current = table->buckets[i];
        while (current != NULL) {
            total_nodes++;
            current = current->next;
        }
    }
    printf("Sequential hash table has %d nodes\n", total_nodes);
    
    free(element_pool);
    free_hash_table(table);
    
    return elapsed_time;
}

// Device-side hash table structure
struct GPUHashTable {
    int* locks;        // Lock for each bucket
    int* next_indices; // Array to track next available node index
    int* keys;         // Array of keys
    int* values;       // Array of values
    int* next_ptrs;    // Array of next pointers
    int* bucket_heads; // Array of bucket head pointers
    int size;          // Number of buckets
};

// Hash table server kernel message handler
__device__ void process_hash_table_msg(Message* msg, GPUHashTable* ht, int* locks) {
    int bucket = msg->counter_idx;
    
    // Acquire lock for this bucket (using shared memory lock)
    while (atomicCAS(&locks[bucket], 0, 1) != 0) {
    }
    
    // Get next available node index
    int idx = atomicAdd(&ht->next_indices[0], 1);
    
    if (idx < MAX_LIST_NODES) {
        // Store key and value
        ht->keys[idx] = bucket; // Using bucket as key 
        ht->values[idx] = idx;  // Using index as value 
        
        // Update linked list (insert at head)
        int old_head = ht->bucket_heads[bucket];
        ht->next_ptrs[idx] = old_head;
        ht->bucket_heads[bucket] = idx;
    }
    
    // Release the lock
    __threadfence(); 
    atomicExch(&locks[bucket], 0);
}

// Modified server kernel for hash table operations
__global__ void hash_table_server_kernel(int* counters, int num_counters, int num_server_blocks, 
                                        Buffer* bufs, int* done, int num_threads,
                                        GPUHashTable* hash_table) {
    bool is_server = blockIdx.x < num_server_blocks;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (is_server) {
        // Initialize shared memory locks
        extern __shared__ int locks[];
        for (int i = threadIdx.x; i < num_counters; i += blockDim.x) {
            locks[i] = 0;
        }
        
        if (threadIdx.x == 0) {
            printf("Hash Table Server %d initialized locks to 0 in shared memory\n", blockIdx.x);
        }
        
        __syncthreads();
        
        Buffer* my_buf = &bufs[blockIdx.x];
        int empty_iterations = 0;
        const int MAX_EMPTY_ITERATIONS = 1000;
        
        while (true) {
            int sent = atomicAdd(done, 0);
            bool processed_message = false;
            
            // Check for messages in buffer
            Message msg;
            if (dequeue(my_buf, &msg)) {
                process_hash_table_msg(&msg, hash_table, locks);
                processed_message = true;
                empty_iterations = 0;
            } else {
                empty_iterations++;
            }
            
            // Exit condition with safety check - now checking against total client threads
            if (sent >= num_threads) {
                if (isEmpty(my_buf) || empty_iterations > MAX_EMPTY_ITERATIONS) {
                    if (threadIdx.x == 0) {
                        printf("Server %d exiting. done=%d, target=%d, empty_iterations=%d\n", 
                              blockIdx.x, sent, num_threads, empty_iterations);
                    }
                    break;
                }
            }
            
            // Add a small delay if no messages were processed
            if (!processed_message) {
                // Short yield/backoff to reduce contention
                for (int i = 0; i < 10; i++) { 
                    __threadfence_block();
                }
            }
        }
    } else {
        // Client code - one thread per message
        // Calculate the client thread ID (relative to all client threads)
        int client_tid = tid - (num_server_blocks * blockDim.x);
        
        // Only threads within the pool size range participate
        if (client_tid < num_threads) {
            // Generate hash bucket based on the client thread ID
            int counter = client_tid % num_counters;
            int target_server = counter % num_server_blocks;
            
            // Send exactly one message per client thread
            send_msg(target_server, counter, bufs, done);
            
            if (threadIdx.x == 0 && blockIdx.x == num_server_blocks) {
                printf("Client threads started sending messages\n");
            }
        }
    }
    
    __syncthreads();
}

// Run CUDA hash table benchmark
double run_cuda_benchmark(int pool_size, int num_servers, int num_clients) {
    printf("Running CUDA benchmark with pool size %d\n", pool_size);
    
    // Allocate host memory
    int* h_bucket_heads = (int*)malloc(sizeof(int) * HT_SIZE);
    int* h_next_indices = (int*)malloc(sizeof(int));
    h_next_indices[0] = 0; // First available slot
    
    for (int i = 0; i < HT_SIZE; i++) {
        h_bucket_heads[i] = -1; // -1 indicates empty bucket
    }
    
    // Allocate device memory for hash table
    GPUHashTable h_table;
    GPUHashTable* d_table;
    
    CHECK(hipMalloc(&d_table, sizeof(GPUHashTable)));
    CHECK(hipMalloc(&h_table.locks, sizeof(int) * HT_SIZE));
    CHECK(hipMalloc(&h_table.next_indices, sizeof(int)));
    CHECK(hipMalloc(&h_table.keys, sizeof(int) * MAX_LIST_NODES));
    CHECK(hipMalloc(&h_table.values, sizeof(int) * MAX_LIST_NODES));
    CHECK(hipMalloc(&h_table.next_ptrs, sizeof(int) * MAX_LIST_NODES));
    CHECK(hipMalloc(&h_table.bucket_heads, sizeof(int) * HT_SIZE));
    
    h_table.size = HT_SIZE;
    
    // Initialize device memory
    CHECK(hipMemcpy(h_table.next_indices, h_next_indices, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h_table.bucket_heads, h_bucket_heads, sizeof(int) * HT_SIZE, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_table, &h_table, sizeof(GPUHashTable), hipMemcpyHostToDevice));
    
    // Allocate device memory for message passing
    Buffer* d_bufs;
    int* d_done;
    int* d_shared_data;
    
    CHECK(hipMalloc(&d_shared_data, HT_SIZE * sizeof(int)));
    CHECK(hipMemset(d_shared_data, 0, HT_SIZE * sizeof(int)));
    CHECK(hipMalloc(&d_done, sizeof(int)));
    CHECK(hipMemset(d_done, 0, sizeof(int)));
    CHECK(hipMalloc(&d_bufs, num_servers * sizeof(Buffer)));
    
    // Initialize buffers on host and copy to device
    Buffer* h_bufs = (Buffer*)malloc(num_servers * sizeof(Buffer));
    memset(h_bufs, 0, num_servers * sizeof(Buffer));
    CHECK(hipMemcpy(d_bufs, h_bufs, num_servers * sizeof(Buffer), hipMemcpyHostToDevice));
    free(h_bufs);
    
    // Start timer
    printf("Launching kernel with %d server blocks and %d client threads...\n", 
           num_servers, num_clients);
    hipDeviceSynchronize();
    double start_time = get_time();
    
    // Launch kernel
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size(num_servers + (num_clients + block_size.x - 1) / block_size.x);
    int shared_mem_size = HT_SIZE * sizeof(int); // For locks
    
    printf("Grid size: %d, Block size: %d, Shared mem: %d bytes\n", 
           grid_size.x, block_size.x, shared_mem_size);
    
    hash_table_server_kernel<<<grid_size, block_size, shared_mem_size>>>(
        d_shared_data, HT_SIZE, num_servers, d_bufs, d_done, 
        num_clients, d_table);
    
    // Wait for kernel to complete with timeout
    hipError_t error = hipSuccess;
    for (int i = 0; i < 5; i++) {  // Try 5 times with increasing timeouts
        error = hipDeviceSynchronize();
        if (error == hipSuccess) break;
        
        printf("Warning: Synchronize timeout, retrying... (attempt %d/5)\n", i+1);
        usleep(1000000);  // Wait 1 second before retrying (using microseconds)
    }
    
    // End timer
    double end_time = get_time();
    double elapsed_time = end_time - start_time;
    
    if (error != hipSuccess) {
        printf("Error: Kernel execution failed or timed out: %s\n", hipGetErrorString(error));
        // Reset device to recover from errors
        hipDeviceReset();
        return 999.0;  // Return an obviously invalid time
    }
    
    // Copy results back to verify
    int h_next_index;
    CHECK(hipMemcpy(&h_next_index, h_table.next_indices, sizeof(int), hipMemcpyDeviceToHost));
    printf("CUDA hash table has %d nodes\n", h_next_index);
    
    // Free memory
    free(h_bucket_heads);
    free(h_next_indices);
    
    CHECK(hipFree(h_table.locks));
    CHECK(hipFree(h_table.next_indices));
    CHECK(hipFree(h_table.keys));
    CHECK(hipFree(h_table.values));
    CHECK(hipFree(h_table.next_ptrs));
    CHECK(hipFree(h_table.bucket_heads));
    CHECK(hipFree(d_table));
    CHECK(hipFree(d_bufs));
    CHECK(hipFree(d_done));
    CHECK(hipFree(d_shared_data));
    
    return elapsed_time;
}

int main(int argc, char** argv) {
    // Seed random number generator
    srand(time(NULL));
    
    // Default parameters
    int num_servers = 4;           // Number of server blocks
    int collision_factor = CF_1K;  // Default collision factor (1024)
    
    // Using boost program options for argument parsing
    try {
        bpo::options_description desc("Hash Table Benchmark Options");
        desc.add_options()
            ("help", "Show help message")
            ("cf", bpo::value<int>()->default_value(CF_1K), "Collision factor (256, 1024, 32768, or 131072)")
            ("servers", bpo::value<int>()->default_value(4), "Number of server blocks");
        
        bpo::variables_map vm;
        bpo::store(bpo::parse_command_line(argc, argv, desc), vm);
        bpo::notify(vm);
        
        if (vm.count("help")) {
            std::cout << desc << "\n";
            return 1;
        }
        
        collision_factor = vm["cf"].as<int>();
        num_servers = vm["servers"].as<int>();
        
        // Validate collision factor is one of the expected values
        if (collision_factor != CF_256 && collision_factor != CF_1K && 
            collision_factor != CF_32K && collision_factor != CF_128K) {
            std::cerr << "Error: Collision factor must be one of: 256, 1024, 32768, or 131072\n";
            return 1;
        }
        
    } catch (std::exception& e) {
        std::cerr << "Error: " << e.what() << "\n";
        return 1;
    }
    
    // Print benchmark parameters
    std::cout << "Running benchmark with:\n"
              << "  - Collision factor: " << collision_factor << "\n"
              << "  - Number of server blocks: " << num_servers << "\n";
              
    // Number of client threads equals the pool size (collision factor)
    int num_clients = collision_factor;
    
    // Run sequential benchmark
    double seq_time = run_sequential_benchmark(collision_factor);
    
    // Run CUDA benchmark
    double cuda_time = run_cuda_benchmark(collision_factor, num_servers, num_clients);
    
    // Print results
    printf("\nResults summary:\n");
    printf("Sequential time: %.6f seconds\n", seq_time);
    printf("CUDA time: %.6f seconds\n", cuda_time);
    printf("Speedup: %.2fx\n", seq_time / cuda_time);
    
    return 0;
}